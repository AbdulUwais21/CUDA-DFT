#include "hip/hip_runtime.h"
//General Include
#include <stdlib.h>
#include <math.h>
#include <stdio.h>

//CUDA Include(s)
// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <helper\helper_functions.h>
#include <helper\hip/hip_runtime_api.h>
//#include "exception.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

typedef float2 cplx; //Complex Version CUDA - cuFFT
#define N 32 // Try Out
#define M_PI 3.14159265358979323846

//Private Function Prototype
void dft(cplx in[], cplx out[], int n);
__global__ void dft_CUDA(cplx *in, cplx *out, int n);
__device__ double phaseDFT(unsigned int idx, unsigned int i, int n);
__device__ void takeOut(cplx *in, cplx *out, int n);

int main(int argc, char **argv) // int argc, char **argv
{
	
	int signalSize = 0, p = 1;
    if (argc > 1)
    {
    	signalSize = atoi(argv[1]);
    }
    else 
    	signalSize = N;

    //To Fill Array for power of two number array
    while (signalSize>p)
    {
    	p *= 2;
    }

    if (signalSize < p) 
    	{
    		printf("Signal Size = Not Power of Two\n");
    		signalSize = p; //Add to meet the requirement of power of Two
    		printf("Add --SignalSize-- to meet the requirement of power of Two\n");
    		//return 0;
    	}

    printf("Number Input Array : %d\n", signalSize);
    
    cplx *x = (cplx *)malloc(sizeof(cplx)*signalSize);
    cplx *y_h = (cplx *)malloc(sizeof(cplx)*signalSize); 
    memset(y_h, 1, sizeof(cplx)*signalSize);
    cplx *y_d = (cplx *)malloc(sizeof(cplx)*signalSize);
    memset(y_d, 1, sizeof(cplx)*signalSize);

    for (int i = 0; i<(signalSize/2); i++)
    {
    	x[i].x = 1;
    	x[i+(signalSize/2)].x = 0;
    	x[i].y = 0;
    	x[i+(signalSize/2)].y = 0;
    }

    	printf("Before Forward FFT\n");
    for (int j = 0; j<signalSize; j++)
    {
        y_d[j].x = 0;
        y_d[j].y = 0;
    	printf("Starting values: X = %.2f %+.2fi\tY  = %.2f %+.2fi\n", x[j].x, x[j].y, y_h[j].x, y_h[j].y);
    }

    //Initializing Number Thread and Block
    int maxThreads = 32;
    int n_threads_per_blocks = maxThreads;
    int numBlock = signalSize / n_threads_per_blocks; //Initial Number Block
    //int pKali = 1;

    printf("Number Threads / Block = %d\n", n_threads_per_blocks);
    printf("Number Block = %d\n", numBlock);
    //Calculate using DFT - Non CUDA
    dft(x, y_h, signalSize); //Out-Place ; X Input, Y Output, n Number Array Input

    printf("After Forward DFT\n");
    for (int j = 0;j<signalSize;j++)
    {
    	printf("Result values: Initial Signals X  = %.2f %+.2fi \t Y from DFT = %.2f %+.2fi\n", x[j].x, x[j].y, y_h[j].x, y_h[j].y);	
    }

    //CUDA Malloc
    cplx *d_x;
    checkCudaErrors(hipMalloc((void **)&d_x, sizeof(cplx)*signalSize));
    checkCudaErrors(hipMemset(d_x, 1, sizeof(cplx)*signalSize));
    cplx *d_y;
    checkCudaErrors(hipMalloc((void **)&d_y, sizeof(cplx)*signalSize*signalSize));
    checkCudaErrors(hipMemset(d_y, 1, sizeof(cplx)*signalSize*signalSize));
    // cplx *d_temp;
    // checkCudaErrors(hipMalloc((void **)&d_temp, sizeof(cplx)*signalSize*signalSize));
    // checkCudaErrors(hipMemset(d_temp, 0, sizeof(cplx)*signalSize*signalSize));

    //CUDA Memcpy
    checkCudaErrors(hipMemcpy(d_x, x, sizeof(cplx)*signalSize, hipMemcpyHostToDevice));

    //Launch Kernel CUDA ----- In-Place ; X Input, Y Output, n Number Array Input
    dft_CUDA<<<numBlock, n_threads_per_blocks>>>(d_x, d_y, signalSize);
    //hipDeviceSynchronize();
    // hipDeviceSynchronize();

    //CUDA Memcpy
    checkCudaErrors(hipMemcpy(y_d, d_x, sizeof(cplx)*signalSize, hipMemcpyDeviceToHost));

    
    printf("After Forward DFT - CUDA\n");

    printf("\tFROM DFT \t\t\t\t\t\t\t FROM DFT CUDA\n");

    cplx diff;
    for (int j = 0;j<signalSize;j++)
    {
        diff.x = y_h[j].x - y_d[j].x;
        diff.y = y_h[j].y - y_d[j].y;
    	printf("Result values: Initial Signals Y Host  = %.5f %+.5fi \t Y from DFT CUDA = %.5f %+.5fi >>> Difference = %.5f %+.5fi\n", y_h[j].x, y_h[j].y, y_d[j].x, y_d[j].y, diff.x, diff.y);	
    }

    //Free Memory - CPU & CUDA Arch.
    free(x);
    free(y_h);
    free(y_d);
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));

    getchar();
    return 0;
}

//Out-Place Algorithm and Arithmetic Complexity O(n^2)   
void dft(cplx in[], cplx out[], int n)
{

	double sumr = 0, sumi = 0;
	double phase = 0.0;
	for (int s =0; s<n; s++)  //Loop Output
	{
		sumr = 0.0;
		sumi = 0.0;

		for (int t = 0; t < n ; t++) //Loop for Operating Input with DFT Equation
		{
			phase = 2 * M_PI * t * s/n; //M_PI = PHI
			sumr += in[t].x * cos(phase)  + in[t].y * sin(phase);
			sumi += (-1) * in[t].x * sin(phase) + in[t].y * cos(phase);
		}
		out[s].x = sumr;
		out[s].y = sumi;
	}
}

__global__ void dft_CUDA( cplx *in,  cplx *out, int n)
{
    //unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ cplx temp[32];
    unsigned int idx = threadIdx.x;
    double sumr = 0.0,
            sumi = 0.0;
    //unsigned int incrementId = blockDim.x * gridDim.x;
    //double phase;
   // int toIncrement = 0;
    
    for (int i = 0; i < n ; i++)
    {
        out[idx+i*n].x = in[i].x *cos(phaseDFT(idx,i,n)) + in[i].y * sin(phaseDFT(idx, i, n));
        out[idx+i*n].y = (-1) * in[i].x * sin(phaseDFT(idx, i, n)) + in[i].y * cos(phaseDFT(idx, i, n));
        printf("out[%u+%d*%d = %d] = %.2f +%.2fi\n",idx, i, n, (idx+i*n), out[idx+i*n].x, out[idx+i*n].y);
        printf("Phase = %.2f", phaseDFT(idx,i,n));
        __syncthreads();//+ in[i].y * sin(2*M_PI*idx*i);

        // in[idx+n].x = out[idx*n+i].x;
        // in[idx+n].y = out[idx*n+i].y;
        // printf("out Result[%u*%d+%d = %d] = %.2f +%.2fi\n",idx, n, i, (i+idx*n), out[i+idx*n].x, out[i+idx*n].y);
      

    }

      for (int j = 0; j < n; j++)
            {
                temp[idx].x = out[idx*n+j].x;
                temp[idx].y = out[idx*n+j].y;
                printf("out Result[%u*%d+%d = %d] = %.2f +%.2fi\n",idx, n, j, (j+idx*n), out[j+idx*n].x, out[j+idx*n].y);
                __syncthreads();

                sumr = 0.0;
                sumi = 0.0;
                for (int k = 0; k < n; k++)
                {
                    sumr = sumr + temp[k].x;
                    sumi = sumi + temp[k].y;
                    printf("in[%d] Result = %.2f +%.2fi\n",j, temp[k].x, temp[k].y);
                }

                in[j].x = sumr;
                in[j].y = sumi;
            }


    //takeOut(out, in, n);

    // for (int k = 0; k < n ; i++)
    // {
    //     for (int j = 0; j < n; j++)
    //     {
    //         sumr = sumr + out[i+j*n].x;
    //         printf("SUmr = %.2f + out[%u + %d * %d].x = Total Idx = %d \n", sumr, i, j, n, j*n+i);
    //         sumi = sumi + out[i+j*n].y;
    //         printf("out Result[%u+%d*%d = %d] = %.2f +%.2fi\n",i, j, n, (i+j*n), out[i+j*n].x, out[i+j*n].y);

           
    //         printf("in[%d] Result[%u+%d*%d = %d] = %.2f +%.2fi\n",i, i, j, n, (i+j*n), in[i+j*n].x, in[i+j*n].y);
    //     }
    //     in[i].x = sumr;
    //     in[i].y = sumi;
    // }

    // for (int i = 0; i < n; i++)
    // {   
    //     in[idx].x = in[idx].x + out[i+idx*n].x;
    //     printf("SUmr = %.2f + out[%u + %d * %d].x = Total Idx = %d \n", sumr, idx, i, n, idx*n+i);
    //     in[idx].y = in[idx].y + out[i+idx*n].y;
    //     printf("out Result[%u+%d*%d = %d] = %.2f +%.2fi\n",idx, i, n, (i+idx*n), out[i+idx*n].x, out[i+idx*n].y);

    //     // in[i].x = sumr;
    //     // in[i].y = sumi;
    //     printf("in[%d] Result[%u+%d*%d = %d] = %.2f +%.2fi\n",i, idx, i, n, (i+idx*n), in[i+idx*n].x, in[i+idx*n].y);
    // }
   // shiftIn(out,n,(-1*toIncrement));
}

__device__ double phaseDFT(unsigned int idx, unsigned int i, int n)
{
    double out;
    out = 2 * M_PI * idx * i / n;
    return out;
}

__device__ void takeOut(cplx *in, cplx *out, int n)
{
    double sumr = 0.0, sumi = 0.0;
    for (int i = 0; i < n ; i++)
    {
        for (int j = 0; j < n; j++)
        {
            sumr = sumr + in[i+j*n].x;
            printf("SUmr = %.2f + out[%u + %d * %d].x = Total Idx = %d \n", sumr, i, j, n, j*n+i);
            sumi = sumi + in[i+j*n].y;
            printf("out Result[%u+%d*%d = %d] = %.2f +%.2fi\n",i, j, n, (i+j*n), out[i+j*n].x, out[i+j*n].y);

            printf("in[%d] Result[%u+%d*%d = %d] = %.2f +%.2fi\n",i, i, j, n, (i+j*n), in[i+j*n].x, in[i+j*n].y);
        }
        out[i].x = sumr;
        out[i].y = sumi;
    }
}