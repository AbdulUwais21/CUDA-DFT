#include "hip/hip_runtime.h"
//General Include
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <sys/time.h>

//CUDA Include(s)
// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <helper/helper_functions.h>
#include <helper/hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

typedef float2 cplx; //Complex Version CUDA - cuFFT
#define N 16 // Try Out
#define M_PI 3.14159265358979323846
#define NStreams 16

//Private Function Prototype
void dft(cplx in[], cplx out[], int n);
void fft_it(cplx In[], int u);
unsigned int bitrev( unsigned int n,  unsigned int bits) ;
// __global__ void dft_CUDA(cplx *in, cplx *out, int n);
__global__ void dft_CUDA( cplx *in,  cplx *out, int n, int n_block_in_stream, int n_stream, int startIdx, int outIdx);
__global__ void sumDFT_CUDA(cplx *in, cplx *out, int n_stream);
__device__ double phaseDFT(unsigned int idx, unsigned int i, int n);

int main(int argc, char **argv) // int argc, char **argv
{
	//Initialising
    hipError_t result;
    size_t freeMemGet1, totalMemGet1,
            freeMemGet2, totalMemGet2;
	int signalSize = 0, p = 1;
    clock_t startmal, stopmal,
            startDFTCPU, stopDFTCPU,
            startFFTCPU, stopFFTCPU;
    float cpuTimer1, cpuTimer2, cpuTimer3;

    //For GPU


    if (argc > 1)
    {
    	signalSize = atoi(argv[1]);
    }
    else 
    	signalSize = N;

    //To Fill Array for power of two number array
    while (signalSize>p)
    {
    	p *= 2;
    }

    if (signalSize < p) 
    	{
    		printf("Signal Size = Not Power of Two\n");
    		signalSize = p; //Add to meet the requirement of power of Two
    		printf("Add --SignalSize-- to meet the requirement of power of Two\n");
    		//return 0;
    	}

    printf("Number Input Array : %d\n", signalSize);

    size_t nBytes = signalSize * sizeof(cplx);
    
    startmal = clock();
    ////////////////////////////////////////////////////////
    cplx *x = (cplx *)malloc(sizeof(cplx)*signalSize);
    cplx *y_h = (cplx *)malloc(sizeof(cplx)*signalSize); 
    memset(y_h, 0, sizeof(cplx)*signalSize);
    cplx *y_d = (cplx *)malloc(sizeof(cplx)*signalSize);
    memset(y_d, 0, sizeof(cplx)*signalSize);

    // cplx *x;  
    // checkCudaErrors(hipHostAlloc((void **)&x, nBytes, hipHostMallocDefault));
    // cplx *y_h;
    // checkCudaErrors(hipHostAlloc((void **)&y_h, nBytes, hipHostMallocDefault));
    // memset(y_h, 0, sizeof(cplx)*signalSize);
    // cplx *y_d;
    // checkCudaErrors(hipHostAlloc((void **)&y_d, nBytes, hipHostMallocDefault));
    // memset(y_d, 0, sizeof(cplx)*signalSize);

    for (int i = 0; i<(signalSize/2); i++)
    {
    	x[i].x = 1;
    	x[i+(signalSize/2)].x = 0;
    	x[i].y = 0;
    	x[i+(signalSize/2)].y = 0;
    }

    ////////////////////////////////////////////////////////
    stopmal = clock();
    cpuTimer1 = (float)(stopmal - startmal) / (CLOCKS_PER_SEC);

    	printf("Before Forward FFT\n");
    // for (int j = 0; j<signalSize; j++)
    // {
    //     // y_d[j].x = 0;
    //     // y_d[j].y = 0;
    // 	printf("Starting values: X = %.2f %+.2fi\tY  = %.2f %+.2fi\n", x[j].x, x[j].y, y_h[j].x, y_h[j].y);
    // }

    //Initializing Number Thread and Block
    int WarpSize = 32;
    int n_threads_per_blocks = WarpSize;
    //Initial Number Block

    // int numBlock = signalSize / n_threads_per_blocks;
    if (signalSize <= WarpSize)
    {
        n_threads_per_blocks = signalSize;
        // numBlock = 1;
    }

    dim3 block (n_threads_per_blocks);
    dim3 grid ((signalSize + block.x - 1)/ block.x);
    printf("<<< Grid (%d, %d), Block (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);
    // int n_threads_per_blocks2 = signalSize / n_threads_per_blocks;
    //int pKali = 1;

    /////For Testing 2D Block 1D Grid
    // dim3 block (n_threads_per_blocks, n_threads_per_blocks2);

    // printf("Number Threads / Block = %d\n", n_threads_per_blocks);
    // printf("Number Block = %d\n", numBlock);

    
    /////////////////////////////////////////////////////////////////////////////
    startDFTCPU = clock();
    //Calculate using DFT - Non CUDA
    dft(x, y_h, signalSize); //Out-Place ; X Input, Y Output, n Number Array Input
    stopDFTCPU = clock();
    cpuTimer2 = (float)(stopDFTCPU - startDFTCPU) / (CLOCKS_PER_SEC);
    /////////////////////////////////////////////////////////////////////////////

    // printf("After Forward DFT\n");
    // for (int j = 0;j<signalSize;j++)
    // {
    // 	printf("Result values: Initial Signals X  = %.2f %+.2fi \t Y from DFT = %.2f %+.2fi\n", x[j].x, x[j].y, y_h[j].x, y_h[j].y);	
    // }

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////CUDA///////////////////////////////////////////////////////////
    //Cuda Event
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // int NStreams = 0;
    // if (signalSize < NumStreams)
    // {
    //     NStreams = signalSize;
    // }
    // else
    // {
    //     NStreams = NumStreams;
    // }
    //Determine Number Grid Parallel Operation
    int numElStreams = signalSize / NStreams;
    // size_t elStreamByte = numElStreams * sizeof(cplx);
    // grid.x = (numElStreams + block.x -1)/block.x;

    //CUDA Stream Initializing
    hipStream_t streams[NStreams];

    for (int i = 0; i < NStreams; i++)
    {
        checkCudaErrors(hipStreamCreate(&streams[i]));
    }
    //CUDA Malloc
    cplx *d_x;
    checkCudaErrors(hipMalloc((void **)&d_x, nBytes));
    cplx *d_y;
    checkCudaErrors(hipMalloc((void **)&d_y, nBytes*NStreams));

    //CUDA Memcpy
    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipMemcpy(d_x, x, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float cuda_memcpyH2D_time;
    checkCudaErrors(hipEventElapsedTime(&cuda_memcpyH2D_time, start, stop));
    
    //Launch Kernel CUDA ----- Out-Place ; X Input, Y Output, n Number Array Input
    checkCudaErrors(hipEventRecord(start, 0));
    int j =0;
    for (int i = 0; i < NStreams ; i++)
    {
        int startIdx = i * numElStreams;
        dft_CUDA<<<grid, block, 0, streams[i]>>>(d_x, d_y, signalSize, numElStreams, NStreams, startIdx, j);    
        j++;
    }

    sumDFT_CUDA<<<grid, block>>>(d_y, d_x, NStreams);
    // dft_CUDA<<<grid, block>>>(d_x, d_y, signalSize);    
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float kernel_time;
    checkCudaErrors(hipEventElapsedTime(&kernel_time, start, stop));

    // hipDeviceSynchronize();
    // checkCudaErrors(hipDeviceSynchronize());

    result = hipMemGetInfo(&freeMemGet1, &totalMemGet1);
    if (result == hipSuccess)
    {
    printf("\nAfter Kernel Execution :Available Memory : %d MB, Total Memory : %d MB\n", freeMemGet1/(1024*1024), totalMemGet1/(1024*1024));
    }

    //CUDA Memcpy
    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipMemcpy(y_d, d_x, nBytes, hipMemcpyDeviceToHost));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float cuda_memcpyD2H_time;
    checkCudaErrors(hipEventElapsedTime(&cuda_memcpyD2H_time, start, stop));

    float GPU_total_time = cuda_memcpyH2D_time + kernel_time + cuda_memcpyD2H_time;

    /////////////////////////////////////////////////////////////////////////////////////////////////////
    //Arithmetic Complexity 0(n log2 n)
    //In-Place Algorithm
    startFFTCPU = clock();
    fft_it(x, signalSize);
    stopFFTCPU = clock();
    cpuTimer3 = (float)(stopFFTCPU - startFFTCPU) / CLOCKS_PER_SEC;
    /////////////////////////////////////////////////////////////////////////////////////////////////////


    printf("After Forward DFT - CUDA\n");

    printf("\tFROM DFT CPU \t\t\t\t\t\t\t FROM DFT CUDA\n");

    
    /////////////////////Result Check//////////////////////////////////
    if ( signalSize < 512)
    {
        cplx diff;
        for (int j = 0;j<signalSize;j++)
        {
            diff.x = y_h[j].x - y_d[j].x;
            diff.y = y_h[j].y - y_d[j].y;
        	printf("Result values: Signals FFT CPU = %.5f %+.5fi \t Signals Y from DFT CPU Host  = %.5f %+.5fi \t Y from DFT CUDA = %.5f %+.5fi \t\t>>> Difference = %.5f %+.5fi\n", x[j].x, x[j].y, y_h[j].x, y_h[j].y, y_d[j].x, y_d[j].y, diff.x, diff.y);	
        }
    }
    //////////////////CALCULATING - Check Memory GPU////////////////////////////////
    result = hipMemGetInfo(&freeMemGet2, &totalMemGet2);
    if (result == hipSuccess)
    {
    printf("\nAvailable Memory : %d MB, Total Memory : %d MB\n", freeMemGet2/(1024*1024), totalMemGet2/(1024*1024));
    }

    /////////////////CPU TIMER////////////////////////
    printf("\nCPU Malloc, Memset, and Initializing = %f ms\n", cpuTimer1*1000);
    printf("CPU DFT Function = %f ms\n", cpuTimer2*1000);
    printf("CPU FFT Iterative Function = %f ms\n", cpuTimer3*1000);

     printf("\n");
    printf("Measured Timing and Throughput :\n");
    printf("Memcpy H2D : \t %f ms (%f GB/s)\n", cuda_memcpyH2D_time, nBytes/cuda_memcpyH2D_time);
    printf("Kernel  : \t %f ms (%f GB/s)\n", kernel_time, nBytes/kernel_time);
    printf("Memcpy D2H : \t %f ms (%f GB/s)\n", cuda_memcpyD2H_time, nBytes/cuda_memcpyH2D_time);
    printf("GPU Total Time: \t %f ms (%f GB/s)\n", GPU_total_time, nBytes/GPU_total_time);


    //Free Memory - CPU & CUDA Arch.
    free(x);
    free(y_h);
    free(y_d);
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    for (int i = 0; i < NStreams; i++)
    {
        checkCudaErrors(hipStreamDestroy(streams[i]));
    }
    checkCudaErrors(hipDeviceReset());

    getchar();
    return 0;
}

//Out-Place Algorithm and Arithmetic Complexity O(n^2)   
void dft(cplx in[], cplx out[], int n)
{

	double sumr = 0, sumi = 0;
	double phase = 0.0;
	for (int s =0; s<n; s++)  //Loop Output
	{
		sumr = 0.0;
		sumi = 0.0;

		for (int t = 0; t < n ; t++) //Loop for Operating Input with DFT Equation
		{
			phase = 2 * M_PI * t * s/n; //M_PI = PHI
			sumr += in[t].x * cos(phase)  + in[t].y * sin(phase);
			sumi += (-1) * in[t].x * sin(phase) + in[t].y * cos(phase);
		}
		out[s].x = sumr;
		out[s].y = sumi;
	}
}

//Arithmetic Complexity 0(n log2 n)
//In-Place Algorithm
void fft_it(cplx In[], int u)
{
     int n= 1,h_idx= 1, j= 1, k= 1,s= 1, jumlah_bits = 0;
     double *costri = (double *) malloc(sizeof(double)*u/2);
     double *sintri = (double *) malloc(sizeof(double)*u/2);
     cplx *out = (cplx *)malloc(sizeof(cplx)*u);

    n = u;    // Jumlah Data
    int in_bit= n- 1; // Jika data dihitung dari 0
    while (in_bit>=h_idx)
    {
        if (n < 3)
        {
            jumlah_bits = 1;
        }
        else
        {
            h_idx *= 2;
            jumlah_bits += 1;
        }
    }


    for (int i = 0; i<n; i++)
    {
        out[i] = In[i];
    }
    for (int f=0;f<n;f++)
    {
        unsigned int bitr = bitrev(f, jumlah_bits);
        In[f] = out[bitr];
    }

    for (int  m = 0; m < n/2 ; m++)
    {
        costri[m] = cos(2 * M_PI * m / n);
        sintri[m] = sin(2 * M_PI * m / n);
    }

    for (s=2; s<=n ; s*=2)
    {
        int hs = s / 2;
        int ts = n / s;
        
        for (int loop1 = 0; loop1 < n; loop1+=s)
        {
            j = 0;
            k = 0;
            for (j = loop1, k = 0; j < loop1+hs; j++, k += ts)
            {
                double temptre = In[j+hs].x * costri[k] + In[j+hs].y * sintri[k];
                double temptim = (-1) * In[j+hs].x * sintri[k] + In[j+hs].y * costri[k];
                In[j+hs].x = In[j].x - temptre;
                In[j+hs].y = In[j].y - temptim;
                In[j].x += temptre;
                In[j].y += temptim;
            }
        }
    }
}

 unsigned int bitrev( unsigned int n,  unsigned int bits) 
{                           // n = 
     int i, nrev, M;   // nrev will store the bit-reversed pattern
    M = 1<<bits;            // find N: shift left 1 by the number of bits
    nrev = n;   
    for(i=1; i<bits; i++)
    {
        n >>= 1;
        nrev <<= 1;
        nrev |= n & 1;   // give LSB of n to nrev
    }
    nrev &= M-1;         // clear all bits more significant than N-1
    
    return nrev;
}

// __global__ void dft_CUDA( cplx *in,  cplx *out, int n)
// {
//     unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

//     if (idx < n)
//     {
//         for (int i = 0; i < n ; i++ )
//         {
//             out[idx+n].x = (float)((in[i].x *cos(phaseDFT(idx,i,n)) + in[i].y * sin(phaseDFT(idx, i, n))));
//             out[idx+n].y = (float)(((-1) * in[i].x * sin(phaseDFT(idx, i, n)) + in[i].y * cos(phaseDFT(idx, i, n))));
//             out[idx].x = out[idx].x + out[idx+n].x;
//             out[idx].y = out[idx].y + out[idx+n].y;
//         }    
//     }
// }

__global__ void dft_CUDA( cplx *in,  cplx *out, int n, int n_block_in_stream, int n_stream, int startIdx, int outIdx)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int bdimx = blockDim.x;
    float sumr = 0.0, sumi = 0.0;
    // if (idx < n)
    // {
        for (int i = startIdx; i < startIdx+n_block_in_stream ; i++ )
        {
            sumr += (float)((in[i].x *cos(phaseDFT(idx,i,n)) + in[i].y * sin(phaseDFT(idx, i, n))));
            sumi += (float)(((-1) * in[i].x * sin(phaseDFT(idx, i, n)) + in[i].y * cos(phaseDFT(idx, i, n))));
            printf("Idx = %d\n", idx);
        } 
            out[idx+outIdx*bdimx].x = sumr;
            out[idx+outIdx*bdimx].y = sumi;
            printf("Out[%d + %d * %d = %d] = %f + %+fi\n", idx, outIdx, bdimx, idx+(outIdx*bdimx),
                out[idx+outIdx*bdimx].x, out[idx+outIdx*bdimx].y);
    // }

    // if (outIdx == n_block_in_stream)
    // {

    // }
}

__global__ void sumDFT_CUDA(cplx *in, cplx *out, int n_stream)
{
     unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
     unsigned int bdimx = blockDim.x;
     float sumr = 0.0, sumi = 0.0;
    // if (idx < n_stream)
    // {
        for (int i = 0; i < n_stream; i++)
        {
            sumr += in[idx+i*bdimx].x;
            sumi += in[idx+i*bdimx].y;
            printf("In[%d + %d * %d = %d] = %f + %+fi\n", idx, i, bdimx, idx+(i*bdimx),
                in[idx+i*bdimx].x, in[idx+i*bdimx].y);
        }

        out[idx].x = sumr;
        out[idx].y = sumi;
    // }
}


__device__ double phaseDFT(unsigned int idx, unsigned int i, int n)
{
    double out;
    out = 2 * M_PI * idx * i / n;
    return out;
}